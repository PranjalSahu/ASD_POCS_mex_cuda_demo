#include "hip/hip_runtime.h"
#include <math.h>
#include <malloc.h>
#define ABS(a) (a>0?a:-(a))
#define MAX(a,b) (a>b?a:b)
#define MIN(a,b) (a<b?a:b)

#define BLOCK_SIZE_x 16
#define BLOCK_SIZE_y 16

const float eps=1e-8;

extern "C" void Atx_cone_mf_gpu_new(float *X,float *y,float *sc,float cos_phi,float sin_phi,float *y_det,float *z_det,
	float SO,float OD,float scale,float dy_det,float dz_det,float dz,int nx,int ny,int nz,int na,int nb);


inline __device__ float find_l(float x1_0,float y1_0,float x2_0,float y2_0,float dx,float dy,float x,float y)
{   
	float l=0,dx2,dy2,a,b,slope,tmp,tmp2,xi[2],yi[2],x1,y1;
	int i;

	a=x2_0-x1_0;
	b=y2_0-y1_0;
	dx2=dx/2.0f;
	dy2=dy/2.0f;

	if(a==0)
	{   
		tmp=ABS(x1_0-x);
		if(tmp<=dx2){l=dy;}
	}
	else
	{   
		if(b==0)
		{   
			tmp=ABS(y1_0-y);
		if(tmp<=dy2)
		{
			l=dx;
		}
	}
	else
	{   
		x1=x1_0-x;y1=y1_0-y;
		i=0;
		if(ABS(a)>ABS(b))
		{   
			slope=b/a;
			tmp=slope*(-x1)+y1;
			tmp2=slope*dx2;
			if(ABS(tmp-tmp2)<=dy2)
			{
				xi[i]=-dx2;yi[i]=tmp-tmp2;i++;
			}
			if(ABS(tmp+tmp2)<=dy2)
			{
				xi[i]=dx2;yi[i]=tmp+tmp2;i++;
			}

			if(i<2)
			{   
				slope=a/b;
				tmp=slope*(-y1)+x1;
				tmp2=slope*dy2;
				if(ABS(tmp-tmp2)<=dx2)
				{
					yi[i]=-dy2;xi[i]=tmp-tmp2;i++;
				}
				if(i<2)
				{   
					if(ABS(tmp+tmp2)<=dx2)
					{
					yi[i]=dy2;xi[i]=tmp+tmp2;i++;
					}
				}
			}
	}
	else
	{   
		slope=a/b;
		tmp=slope*(-y1)+x1;
		tmp2=slope*dy2;
		if(ABS(tmp-tmp2)<=dx2)
		{
			yi[i]=-dy2;xi[i]=tmp-tmp2;i++;
		}
		if(ABS(tmp+tmp2)<=dx2)
		{
			yi[i]=dy2;xi[i]=tmp+tmp2;i++;
		}

		if(i<2)
		{   
			slope=b/a;
			tmp=slope*(-x1)+y1;
			tmp2=slope*dx2;
			if(ABS(tmp-tmp2)<=dy2)
			{
				xi[i]=-dx2;yi[i]=tmp-tmp2;i++;
			}
			if(i<2)
			{   
				if(ABS(tmp+tmp2)<=dy2)
				{
					xi[i]=dx2;yi[i]=tmp+tmp2;i++;
				}
			}
		}
		}

		if(i==2)
		{   tmp=xi[1]-xi[0];tmp2=yi[1]-yi[0];
		l=(float)sqrt(tmp*tmp+tmp2*tmp2);
		}
		}
	}
	return l;
}


inline __device__ float find_l_3d(float x1_0,float y1_0,float z1_0,float x2_0,float y2_0,float z2_0,float dx,float dy,float dz,float x,float y,float z)
	// assuming c~=0
	// A method for computing the intersecting length of a voxel with a infinitely-narrow beam
	// A better formula will be supplied to improve the speed.
{   
	float l=0,dx2,dy2,dz2,a,b,c,slope,tmp[2],tmp2[2],tmpx,tmpy,tmpz,xi[2],yi[2],zi[2],x1,y1,z1;
	int i;

	a=x2_0-x1_0;b=y2_0-y1_0;c=z2_0-z1_0;
	dx2=dx/2.0f;dy2=dy/2.0f;dz2=dz/2.0f;

	if(a==0)
	{l=find_l(y1_0,z1_0,y2_0,z2_0,dy,dz,y,z);}
	else
	{   if(b==0)
		{l=find_l(x1_0,z1_0,x2_0,z2_0,dx,dz,x,z);}
	else
	{   x1=x1_0-x;y1=y1_0-y;z1=z1_0-z;
	//            x2=x2_0-x;y2=y2_0-y;z2=z2_0-z;

	i=0;
	if(ABS(a)>ABS(b))
	{   slope=b/a;tmp[0]=slope*(-x1)+y1;tmp2[0]=slope*dx2;
	slope=c/a;tmp[1]=slope*(-x1)+z1;tmp2[1]=slope*dx2;
	if(ABS(tmp[0]-tmp2[0])<=dy2&&ABS(tmp[1]-tmp2[1])<=dz2)
	{xi[i]=-dx2;yi[i]=tmp[0]-tmp2[0];zi[i]=tmp[1]-tmp2[1];i++;}
	if(ABS(tmp[0]+tmp2[0])<=dy2&&ABS(tmp[1]+tmp2[1])<=dz2)
	{xi[i]=dx2;yi[i]=tmp[0]+tmp2[0];zi[i]=tmp[1]+tmp2[1];i++;}

	if(i<2)
	{   slope=a/b;tmp[0]=slope*(-y1)+x1;tmp2[0]=slope*dy2;
	slope=c/b;tmp[1]=slope*(-y1)+z1;tmp2[1]=slope*dy2;
	if(ABS(tmp[0]-tmp2[0])<=dx2&&ABS(tmp[1]-tmp2[1])<=dz2)
	{xi[i]=tmp[0]-tmp2[0];yi[i]=-dy2;zi[i]=tmp[1]-tmp2[1];i++;}
	if(i<2)
	{   if(ABS(tmp[0]+tmp2[0])<=dx2&&ABS(tmp[1]+tmp2[1])<=dz2)
	{xi[i]=tmp[0]+tmp2[0];yi[i]=dy2;zi[i]=tmp[1]+tmp2[1];i++;}
	}
	}

	if(i<2)
	{   slope=a/c;tmp[0]=slope*(-z1)+x1;tmp2[0]=slope*dz2;
	slope=b/c;tmp[1]=slope*(-z1)+y1;tmp2[1]=slope*dz2;
	if(ABS(tmp[0]-tmp2[0])<=dx2&&ABS(tmp[1]-tmp2[1])<=dy2)
	{xi[i]=tmp[0]-tmp2[0];yi[i]=tmp[1]-tmp2[1];zi[i]=-dz2;i++;}
	if(i<2)
	{   if(ABS(tmp[0]+tmp2[0])<=dx2&&ABS(tmp[1]+tmp2[1])<=dy2)
	{xi[i]=tmp[0]+tmp2[0];yi[i]=tmp[1]+tmp2[1];zi[i]=dz2;i++;}
	}
	}
	}
	else
	{   slope=a/b;tmp[0]=slope*(-y1)+x1;tmp2[0]=slope*dy2;
	slope=c/b;tmp[1]=slope*(-y1)+z1;tmp2[1]=slope*dy2;
	if(ABS(tmp[0]-tmp2[0])<=dx2&&ABS(tmp[1]-tmp2[1])<=dz2)
	{xi[i]=tmp[0]-tmp2[0];yi[i]=-dy2;zi[i]=tmp[1]-tmp2[1];i++;}
	if(ABS(tmp[0]+tmp2[0])<=dx2&&ABS(tmp[1]+tmp2[1])<=dz2)
	{xi[i]=tmp[0]+tmp2[0];yi[i]=dy2;zi[i]=tmp[1]+tmp2[1];i++;}

	if(i<2)
	{   slope=b/a;tmp[0]=slope*(-x1)+y1;tmp2[0]=slope*dx2;
	slope=c/a;tmp[1]=slope*(-x1)+z1;tmp2[1]=slope*dx2;
	if(ABS(tmp[0]-tmp2[0])<=dy2&&ABS(tmp[1]-tmp2[1])<=dz2)
	{xi[i]=-dx2;yi[i]=tmp[0]-tmp2[0];zi[i]=tmp[1]-tmp2[1];i++;}
	if(i<2)
	{   if(ABS(tmp[0]+tmp2[0])<=dy2&&ABS(tmp[1]+tmp2[1])<=dz2)
	{xi[i]=dx2;yi[i]=tmp[0]+tmp2[0];zi[i]=tmp[1]+tmp2[1];i++;}
	}
	}

	if(i<2)
	{   slope=a/c;tmp[0]=slope*(-z1)+x1;tmp2[0]=slope*dz2;
	slope=b/c;tmp[1]=slope*(-z1)+y1;tmp2[1]=slope*dz2;
	if(ABS(tmp[0]-tmp2[0])<=dx2&&ABS(tmp[1]-tmp2[1])<=dy2)
	{xi[i]=tmp[0]-tmp2[0];yi[i]=tmp[1]-tmp2[1];zi[i]=-dz2;i++;}
	if(i<2)
	{   if(ABS(tmp[0]+tmp2[0])<=dx2&&ABS(tmp[1]+tmp2[1])<=dy2)
	{xi[i]=tmp[0]+tmp2[0];yi[i]=tmp[1]+tmp2[1];zi[i]=dz2;i++;}
	}
	}
	}

	if(i==2)
	{   tmpx=xi[1]-xi[0];tmpy=yi[1]-yi[0];tmpz=zi[1]-zi[0];
	l=(float)sqrt(tmpx*tmpx+tmpy*tmpy+tmpz*tmpz);
	}
	}
	}
	return l;
}




__global__ void Atx_cone_mf_gpu_new_kernel(float *x,float *y,float *sc,float cos_phi,float sin_phi,float *y_det,float *z_det,
	float SO,float OD,float scale,float dy_det,float dz_det,float dz,int nx,int ny,int nz,int na,int nb)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int ix=bx*BLOCK_SIZE_x+tx;
	int iy2=by*BLOCK_SIZE_y+ty;

	if(ix<nx&&iy2<ny*nz)
	{   
		int nx2,ny2,nz2,na2,nb2,ia,ib,iy,iz,na_min,na_max,nb_min,nb_max,idx;
		float xc,yc,zc,xr,yr,SD,l,tmp,x1,y1,z1,x2,y2,z2,d;

		SD=SO+OD;
		na2=na/2;nb2=nb/2;
		nx2=nx/2;ny2=ny/2;nz2=nz/2;
		d=(float)sqrt((1+dz*dz)/2);

		iz=(int)floor((float)iy2/(float)ny);
		iy=iy2-iz*ny;
		idx=iz*ny*nx+iy*nx+ix;

		zc=(float)(iz+0.5-nz2)*dz;
		yc=(float)(iy+0.5-ny2);
		xc=(float)(ix+0.5-nx2);


		xr=cos_phi*xc+sin_phi*yc;
		yr=-sin_phi*xc+cos_phi*yc;

		tmp=SD/((xr+SO)*dy_det);
		na_max=(int)floor((yr+1)*tmp+na2);
		na_min=(int)floor((yr-1)*tmp+na2);

		tmp=SD/((xr+SO)*dz_det);
		nb_max=(int)floor((zc+d)*tmp+nb2);
		nb_min=(int)floor((zc-d)*tmp+nb2);

		for(ib=MAX(0,nb_min);ib<=MIN(nb_max,nb-1);ib++)
		{   
			for(ia=MAX(0,na_min);ia<=MIN(na_max,na-1);ia++)
			{   
				x1=cos_phi*(-SO);
				y1=sin_phi*(-SO);
				z1=0.0;
				x2=cos_phi*OD-sin_phi*y_det[ia];
				y2=sin_phi*OD+cos_phi*y_det[ia];
				z2=z_det[ib];
				l=find_l_3d(x1,y1,z1,x2,y2,z2,1.0,1.0,dz,xc,yc,zc);
				x[idx]+=l*y[ib*na+ia];
				sc[idx]+=l;
			}
		}
	 }
}

__global__ void set2zero(float *x,int nx,int nyz)
{	int bx=blockIdx.x;
int by=blockIdx.y;
int tx=threadIdx.x;
int ty=threadIdx.y;

int ix=bx*BLOCK_SIZE_x+tx;
int iy=by*BLOCK_SIZE_y+ty;

if(ix<nx&&iy<nyz)
{x[iy*nx+ix]=0;}
}

__global__ void scalex(float *x,int nx,int nyz,float scale)
{	int bx=blockIdx.x;
int by=blockIdx.y;
int tx=threadIdx.x;
int ty=threadIdx.y;

int ix=bx*BLOCK_SIZE_x+tx;
int iy=by*BLOCK_SIZE_y+ty;

if(ix<nx&&iy<nyz)
{x[iy*nx+ix]*=scale;}
}

extern "C" void Atx_cone_mf_gpu_new(float *X,float *y,float *sc,float cos_phi,float sin_phi,float *y_det,float *z_det,
	float SO,float OD,float scale,float dy_det,float dz_det,float dz,int nx,int ny,int nz,int na,int nb)
{   
	float *x_d,*y_d,*sc_d,*y_det_d,*z_det_d;
	int nd,N;

	N=nx*ny*nz;
	nd=na*nb;

	hipMalloc(&y_d,nd*sizeof(float));
	hipMalloc(&x_d,N*sizeof(float));
	hipMalloc(&sc_d,N*sizeof(float));
	hipMalloc(&y_det_d,na*sizeof(float));hipMemcpy(y_det_d,y_det,na*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc(&z_det_d,nb*sizeof(float));hipMemcpy(z_det_d,z_det,nb*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(y_d,y,nd*sizeof(float),hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_t((nx+dimBlock.x-1)/dimBlock.x,(ny*nz+dimBlock.y-1)/dimBlock.y);

	set2zero<<<dimGrid_t, dimBlock>>>(x_d,nx,ny*nz);
	set2zero<<<dimGrid_t, dimBlock>>>(sc_d,nx,ny*nz);

	
	Atx_cone_mf_gpu_new_kernel<<<dimGrid_t, dimBlock>>>(x_d,y_d,sc_d,cos_phi,sin_phi,y_det_d,z_det_d,
		SO,OD,scale,dy_det,dz_det,dz,nx,ny,nz,na,nb);
	hipDeviceSynchronize();
	scalex<<<dimGrid_t, dimBlock>>>(x_d,nx,ny*nz,scale);
	scalex<<<dimGrid_t, dimBlock>>>(sc_d,nx,ny*nz,scale);
	hipMemcpy(X,x_d,N*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(sc,sc_d,N*sizeof(float),hipMemcpyDeviceToHost);

	hipFree(x_d);hipFree(y_d);hipFree(sc_d);hipFree(y_det_d);hipFree(z_det_d);
}




extern "C" void Ax_cone_mf_gpu_new(float *X,float *y,float *sr,float cos_phi,float sin_phi,float *y_det,float *z_det,
float SO,float OD,float scale,float dz,int nx,int ny,int nz,int na,int nb);

__global__ void Ax_cone_mf_gpu_kernel_new(float *x,float *y,float *sr,float cos_phi,float sin_phi,float *y_det,float *z_det,
float SO,float OD,float scale,float dz,int nx,int ny,int nz,int na,int nb)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx0=threadIdx.x;
	int ty0=threadIdx.y;

	int ia=bx*BLOCK_SIZE_x+tx0;
	int ib=by*BLOCK_SIZE_y+ty0;

    if(ia<na&&ib<nb)
    {
		int nx2,ny2,nz2,id,ix,iy,iz,cx1,cx2,cy1,cy2,cz1,cz2;
		float x1,y1,x2,y2,z1,z2,xx1,yy1,zz1,xx2,yy2,zz2,slope1,slope2,l,d,tmp,rx,ry,rz;

		nx2=nx/2;
        ny2=ny/2;
        nz2=nz/2;


		id=ib*na+ia;

        x1=cos_phi*(-SO);
        y1=sin_phi*(-SO);
        z1=0.0;
		x2=cos_phi*OD-sin_phi*y_det[ia];
        y2=sin_phi*OD+cos_phi*y_det[ia];
		z2=z_det[ib];

		y[id]=0;
		sr[id]=0;
        // assuming z1-z2 is small
        if(ABS(x1-x2)>ABS(y1-y2))
        {   slope1=(y2-y1)/(x2-x1);
            slope2=(z2-z1)/(x2-x1);
            for(ix=0;ix<nx;ix++)
            {   xx1=(float)(ix-nx2);xx2=xx1+1;
                if(slope1>=0)
                {   yy1=y1+slope1*(xx1-x1)+ny2;
                    yy2=y1+slope1*(xx2-x1)+ny2;
                }
                else
                {   yy1=y1+slope1*(xx2-x1)+ny2;
                    yy2=y1+slope1*(xx1-x1)+ny2;
                }
                cy1=(int)floor(yy1);
                cy2=(int)floor(yy2);
                if(slope2>=0)
                {   zz1=(z1+slope2*(xx1-x1))/dz+nz2;
                    zz2=(z1+slope2*(xx2-x1))/dz+nz2;
                }
                else
                {   zz1=(z1+slope2*(xx2-x1))/dz+nz2;
                    zz2=(z1+slope2*(xx1-x1))/dz+nz2;
                }
                cz1=(int)floor(zz1);
                cz2=(int)floor(zz2);

                if(cy2==cy1)
                {   if(cy1>=0&&cy1<=ny-1)
                    {   if(cz2==cz1)
                        {   if(cz1>=0&&cz1<=nz-1)// 11
                            {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                iy=cy1;iz=cz1;y[id]+=l*x[iz*ny*nx+iy*nx+ix];sr[id]+=l;
                            }
                        }
                        else
                        {   if(cz2>0&&cz2<nz)// 12
                            {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                rz=(cz2-zz1)/(zz2-zz1);
                                iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                iy=cy1;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                            }
                            else
                            {   if(cz2==0)// 13
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rz=(cz2-zz1)/(zz2-zz1);
                                    iy=cy1;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                }
                                if(cz2==nz)// 14
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rz=(cz2-zz1)/(zz2-zz1);
                                    iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                }
                            }
                        }
                    }
                }
                else
                {   if(cy2>0&&cy2<ny)
                    {   if(cz2==cz1)
                        {   if(cz1>=0&&cz1<=nz-1)// 21
                            {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                ry=(cy2-yy1)/d;
                                iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                iy=cy2;iz=cz1;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                            }
                        }
                        else
                        {   if(cz2>0&&cz2<nz)// 22
                            {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                if(ry>rz)
                                {   iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                    iy=cy1;iz=cz2;y[id]+=(ry-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(ry-rz)*l;
                                    iy=cy2;iz=cz2;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                                }
                                else
                                {   iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                    iy=cy2;iz=cz1;y[id]+=(rz-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-ry)*l;
                                    iy=cy2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                }
                            }
                            else
                            {   if(cz2==0)// 23
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(ry>rz)
                                    {   iy=cy1;iz=cz2;y[id]+=(ry-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(ry-rz)*l;
                                        iy=cy2;iz=cz2;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                                    }
                                    else
                                    {   iy=cy2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                    }
                                }
                                if(cz2==nz)// 24
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(ry>rz)
                                    {   iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                    }
                                    else
                                    {   iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                        iy=cy2;iz=cz1;y[id]+=(rz-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-ry)*l;
                                    }
                                }
                            }
                        }
                    }
                    else
                    {   if(cy2==0)
                        {   if(cz2==cz1)
                            {   if(cz1>=0&&cz1<=nz-1)// 31
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;
                                    iy=cy2;iz=cz1;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                                }
                            }
                            else
                            {   if(cz2>0&&cz2<nz)// 32
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(ry>rz)
                                    {   iy=cy2;iz=cz2;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                                    }
                                    else
                                    {   iy=cy2;iz=cz1;y[id]+=(rz-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-ry)*l;
                                        iy=cy2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                    }
                                }
                                else
                                {   if(cz2==0)// 33
                                    {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(ry>rz)
                                        {   iy=cy2;iz=cz2;y[id]+=(1-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-ry)*l;
                                        }
                                        else
                                        {   iy=cy2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                        }
                                    }
                                    if(cz2==nz)// 34
                                    {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(ry>rz)
                                        {
                                        }
                                        else
                                        {   iy=cy2;iz=cz1;y[id]+=(rz-ry)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-ry)*l;
                                        }
                                    }
                                }
                            }
                        }

                        if(cy2==ny)
                        {   if(cz2==cz1)
                            {   if(cz1>=0&&cz1<=nz-1)// 41
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;
                                    iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                }
                            }
                            else
                            {   if(cz2>0&&cz2<nz)// 42
                                {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(ry>rz)
                                    {   iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                        iy=cy1;iz=cz2;y[id]+=(ry-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(ry-rz)*l;
                                    }
                                    else
                                    {   iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                    }
                                }
                                else
                                {   if(cz2==0)// 43
                                    {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(ry>rz)
                                        {   iy=cy1;iz=cz2;y[id]+=(ry-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(ry-rz)*l;
                                        }
                                        else
                                        {
                                        }
                                    }
                                    if(cz2==nz)// 44
                                    {   d=yy2-yy1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        ry=(cy2-yy1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(ry>rz)
                                        {   iy=cy1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                        }
                                        else
                                        {   iy=cy1;iz=cz1;y[id]+=ry*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=ry*l;
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
        else
        {   slope1=(x2-x1)/(y2-y1);
            slope2=(z2-z1)/(y2-y1);
            for(iy=0;iy<ny;iy++)
            {   yy1=(float)(iy-ny2);yy2=yy1+1;
                if(slope1>=0)
                {   xx1=x1+slope1*(yy1-y1)+nx2;
                    xx2=x1+slope1*(yy2-y1)+nx2;
                }
                else
                {   xx1=x1+slope1*(yy2-y1)+nx2;
                    xx2=x1+slope1*(yy1-y1)+nx2;
                }
                cx1=(int)floor(xx1);
                cx2=(int)floor(xx2);
                if(slope2>=0)
                {   zz1=(z1+slope2*(yy1-y1))/dz+nz2;
                    zz2=(z1+slope2*(yy2-y1))/dz+nz2;
                }
                else
                {   zz1=(z1+slope2*(yy2-y1))/dz+nz2;
                    zz2=(z1+slope2*(yy1-y1))/dz+nz2;
                }
                cz1=(int)floor(zz1);
                cz2=(int)floor(zz2);

                if(cx2==cx1)
                {   if(cx1>=0&&cx1<=nx-1)
                    {   if(cz2==cz1)
                        {   if(cz1>=0&&cz1<=nz-1)// 11
                            {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                ix=cx1;iz=cz1;y[id]+=l*x[iz*ny*nx+iy*nx+ix];sr[id]+=l;
                            }
                        }
                        else
                        {   if(cz2>0&&cz2<nz)// 12
                            {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                rz=(cz2-zz1)/(zz2-zz1);
                                ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                ix=cx1;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                            }
                            else
                            {   if(cz2==0)// 13
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rz=(cz2-zz1)/(zz2-zz1);
                                    ix=cx1;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                }
                                if(cz2==nz)// 14
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rz=(cz2-zz1)/(zz2-zz1);
                                    ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                }
                            }
                        }
                    }
                }
                else
                {   if(cx2>0&&cx2<nx)
                    {   if(cz2==cz1)
                        {   if(cz1>=0&&cz1<=nz-1)// 21
                            {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                rx=(cx2-xx1)/d;
                                ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                ix=cx2;iz=cz1;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                            }
                        }
                        else
                        {   if(cz2>0&&cz2<nz)// 22
                            {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                if(rx>rz)
                                {   ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                    ix=cx1;iz=cz2;y[id]+=(rx-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rx-rz)*l;
                                    ix=cx2;iz=cz2;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                                }
                                else
                                {   ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                    ix=cx2;iz=cz1;y[id]+=(rz-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-rx)*l;
                                    ix=cx2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                }
                            }
                            else
                            {   if(cz2==0)// 23
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(rx>rz)
                                    {   ix=cx1;iz=cz2;y[id]+=(rx-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rx-rz)*l;
                                        ix=cx2;iz=cz2;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                                    }
                                    else
                                    {   ix=cx2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                    }
                                }
                                if(cz2==nz)// 24
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(rx>rz)
                                    {   ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                    }
                                    else
                                    {   ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                        ix=cx2;iz=cz1;y[id]+=(rz-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-rx)*l;
                                    }
                                }
                            }
                        }
                    }
                    else
                    {   if(cx2==0)
                        {   if(cz2==cz1)
                            {   if(cz1>=0&&cz1<=nz-1)// 31
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;
                                    ix=cx2;iz=cz1;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                                }
                            }
                            else
                            {   if(cz2>0&&cz2<nz)// 32
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(rx>rz)
                                    {   ix=cx2;iz=cz2;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                                    }
                                    else
                                    {   ix=cx2;iz=cz1;y[id]+=(rz-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-rx)*l;
                                        ix=cx2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                    }
                                }
                                else
                                {   if(cz2==0)// 33
                                    {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(rx>rz)
                                        {   ix=cx2;iz=cz2;y[id]+=(1-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rx)*l;
                                        }
                                        else
                                        {   ix=cx2;iz=cz2;y[id]+=(1-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(1-rz)*l;
                                        }
                                    }
                                    if(cz2==nz)// 34
                                    {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(rx>rz)
                                        {
                                        }
                                        else
                                        {   ix=cx2;iz=cz1;y[id]+=(rz-rx)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rz-rx)*l;
                                        }
                                    }
                                }
                            }
                        }

                        if(cx2==nx)
                        {   if(cz2==cz1)
                            {   if(cz1>=0&&cz1<=nz-1)// 41
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;
                                    ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                }
                            }
                            else
                            {   if(cz2>0&&cz2<nz)// 42
                                {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                    rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                    if(rx>rz)
                                    {   ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                        ix=cx1;iz=cz2;y[id]+=(rx-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rx-rz)*l;
                                    }
                                    else
                                    {   ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                    }
                                }
                                else
                                {   if(cz2==0)// 43
                                    {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(rx>rz)
                                        {   ix=cx1;iz=cz2;y[id]+=(rx-rz)*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=(rx-rz)*l;
                                        }
                                        else
                                        {
                                        }
                                    }
                                    if(cz2==nz)// 44
                                    {   d=xx2-xx1;tmp=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);l=(float)sqrt((d*d+1)*(tmp+(z1-z2)*(z1-z2))/tmp);
                                        rx=(cx2-xx1)/d;rz=(cz2-zz1)/(zz2-zz1);
                                        if(rx>rz)
                                        {   ix=cx1;iz=cz1;y[id]+=rz*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rz*l;
                                        }
                                        else
                                        {   ix=cx1;iz=cz1;y[id]+=rx*l*x[iz*ny*nx+iy*nx+ix];sr[id]+=rx*l;
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
        y[id]*=scale;sr[id]*=scale;
    }
}

extern "C" void Ax_cone_mf_gpu_new(float *X,float *y,float *sr,float cos_phi,float sin_phi,float *y_det,float *z_det,
float SO,float OD,float scale,float dz,int nx,int ny,int nz,int na,int nb)
{   
	float *y_d,*x_d,*sr_d,*y_det_d,*z_det_d;
	int nd,N;

	N=nx*ny*nz;
	nd=na*nb;

	hipMalloc((void**)&y_d,nd*sizeof(float));
	hipMalloc((void**)&x_d,N*sizeof(float));hipMemcpy(x_d,X,N*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc((void**)&sr_d,nd*sizeof(float));
	hipMalloc((void**)&y_det_d,na*sizeof(float));hipMemcpy(y_det_d,y_det,na*sizeof(float),hipMemcpyHostToDevice);
	hipMalloc((void**)&z_det_d,nb*sizeof(float));hipMemcpy(z_det_d,z_det,nb*sizeof(float),hipMemcpyHostToDevice);
	
	//
	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_t((na+dimBlock.x-1)/dimBlock.x,(nb+dimBlock.y-1)/dimBlock.y);
	Ax_cone_mf_gpu_kernel_new<<<dimGrid_t, dimBlock>>>(x_d,y_d,sr_d,cos_phi,sin_phi,y_det_d,z_det_d,
		SO,OD,scale,dz,nx,ny,nz,na,nb);
	hipDeviceSynchronize();
	hipMemcpy(y,y_d,na*nb*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(sr,sr_d,na*nb*sizeof(float),hipMemcpyDeviceToHost);

	
	//
    hipFree(y_d);hipFree(x_d);hipFree(y_det_d);hipFree(z_det_d);hipFree(sr_d);
}



